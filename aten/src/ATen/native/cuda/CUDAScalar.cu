#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/core/Tensor.h>
#include <ATen/Dispatch_v2.h>
#include <ATen/EmptyTensor.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/_local_scalar_dense_native.h>
#endif

#include <ATen/cuda/HIPContext.h>

namespace at::native {

Scalar _local_scalar_dense_cuda(const Tensor& self) {
  Scalar r;
#if !defined(USE_ROCM)
  AT_DISPATCH_V2(
    self.scalar_type(), "_local_scalar_dense_cuda", AT_WRAP([&] {
        // Create pinned memory for the scalar value to avoid implicit
        // locking/sync in cuda library due to pageable memory
        auto value = at::detail::empty_cpu(
          {1}, /* size */
          c10::CppTypeToScalarType<scalar_t>(), /* dtype */
          std::nullopt, /* layout */
          std::nullopt, /* device */
          true, /* pin_memory */
          std::nullopt /* memory format */
        );
        hipStream_t stream = at::cuda::getCurrentCUDAStream();
        at::cuda::memcpy_and_sync((void *)value.const_data_ptr<scalar_t>(), self.const_data_ptr<scalar_t>(), sizeof(scalar_t), hipMemcpyDeviceToHost, stream);
        r = Scalar(*value.const_data_ptr<scalar_t>());
      }), AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX), kComplexHalf, kHalf, kBool, kBFloat16, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));
#else
  // TODO(lufang): Tensor.item() on AMD HIP is not synced in the Recsys models.
  // This is just a short term workaround. Issue is tracked as FBA-388 on the AMD side.
  auto cpu_self = self.cpu();
  AT_DISPATCH_V2(
    self.scalar_type(), "_local_scalar_dense_hip", AT_WRAP([&] {
        r = Scalar(*cpu_self.const_data_ptr<scalar_t>());
      }), AT_EXPAND(AT_ALL_TYPES_AND_COMPLEX), kComplexHalf, kHalf, kBool, kBFloat16, AT_EXPAND(AT_BAREBONES_UNSIGNED_TYPES));

#endif
  return r;
}

} // at::native
